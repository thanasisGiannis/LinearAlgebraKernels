#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <LinearAlgebra/Matrix.hpp>
#include <Orthogonalizations/Cholesky.hpp>

#include <iostream>
#include <memory>

TEST(TestCholesky, Cholesky) 
{

    Orthogonalization::Cholesky<double> cholOrth(100,100);
    EXPECT_EQ(0,0);
}

TEST(TestCholesky, QR)
{

    INT m=200;
    INT n=8;

    /*  A = [ 2 4;
     *        2 5;
     *        2 6;
     *      ]
     */
    std::shared_ptr<LinearAlgebra::Matrix<double>>
    A{new LinearAlgebra::Matrix<double>(m,n)};

    A->rand();

    auto iter = LinearAlgebra::max_element(A->begin(), A->end());
    double normInf = *(iter);

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    C{new LinearAlgebra::Matrix<double>(m,n)};

    for(uint i=0;i<m;i++)
    {
        for(uint j=0;j<n;j++)
        {
            EXPECT_NEAR(0.0,(*C)[i+j*(C->ld())],1e-12*normInf);
        }
    }

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    Q{new LinearAlgebra::Matrix<double>(m,n)};

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    R{new LinearAlgebra::Matrix<double>(n,n)};

    *Q = *A;
    Orthogonalization::Cholesky<double> cholOrth(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::NO_ERROR
              , cholOrth.QR(m,n,Q,R));

    *C = *Q;
    LinearAlgebra::Operation::trmm(LinearAlgebra::Operation::Layout::ColMajor,
                                    LinearAlgebra::Operation::Side::Right,
                                    LinearAlgebra::Operation::Uplo::Upper,
                                    LinearAlgebra::Operation::Op::NoTrans,
                                    LinearAlgebra::Operation::Diag::NonUnit,
                                    m,n,1.0,R->data(), R->ld(),C->data(),C->ld());

    for(uint i=0;i<m;i++)
    {
        for(uint j=0;j<n;j++)
        {
            EXPECT_NEAR(0,std::abs((*A)[i+j*(A->ld())]-(*C)[i+j*(C->ld())]),
                        1e-12*normInf);
        }
    }

    Orthogonalization::Cholesky<double> cholOrthFailed1(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::INVALID_INPUT
              , cholOrthFailed1.QR(n-1,n,Q,R));


    Orthogonalization::Cholesky<double> cholOrthFailed2(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::INVALID_INPUT
              , cholOrthFailed2.QR(m,n,NULL,R));

    Orthogonalization::Cholesky<double> cholOrthFailed3(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::INVALID_INPUT
              , cholOrthFailed3.QR(m,n,Q,NULL));

    R->clear();
    Orthogonalization::Cholesky<double> cholOrthFailed4(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::INVALID_INPUT
              , cholOrthFailed4.QR(m,n,Q,R));

    Q->clear();
    Orthogonalization::Cholesky<double> cholOrthFailed5(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::INVALID_INPUT
              , cholOrthFailed5.QR(m,n,Q,R));

}

