#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <LinearAlgebra/Matrix.hpp>
#include <Orthogonalizations/Cholesky.hpp>

#include <iostream>
#include <memory>

TEST(TestCholesky, Cholesky) 
{

    Orthogonalization::Cholesky<double> cholOrth(100,100);
    EXPECT_EQ(0,0);
}

TEST(TestCholesky, QR)
{

    INT m=1000;
    INT n=21;

    /*  A = [ 2 4;
     *        2 5;
     *        2 6;
     *      ]
     */
    std::shared_ptr<LinearAlgebra::Matrix<double>>
    A{new LinearAlgebra::Matrix<double>(m,n)};

    A->rand();

    auto iter = LinearAlgebra::max_element(A->begin(), A->end());
    double normInf = *(iter);

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    C{new LinearAlgebra::Matrix<double>(m,n)};

    for(uint i=0;i<m;i++)
    {
        for(uint j=0;j<n;j++)
        {
            EXPECT_NEAR(0.0,(*C)[i+j*(C->ld())],1e-12*normInf);
        }
    }

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    Q{new LinearAlgebra::Matrix<double>(m,n)};

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    R{new LinearAlgebra::Matrix<double>(n,n)};

    *Q = *A;
    Orthogonalization::Cholesky<double> cholOrth(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::NO_ERROR
              , cholOrth.QR(m,n,Q,R));

    *C = *Q;
    LinearAlgebra::Operation::trmm(LinearAlgebra::Operation::Layout::ColMajor,
                                    LinearAlgebra::Operation::Side::Right,
                                    LinearAlgebra::Operation::Uplo::Upper,
                                    LinearAlgebra::Operation::Op::NoTrans,
                                    LinearAlgebra::Operation::Diag::NonUnit,
                                    m,n,1.0,R->data(), R->ld(),C->data(),C->ld());

    for(uint i=0;i<m;i++)
    {
        for(uint j=0;j<n;j++)
        {
            EXPECT_NEAR(0,std::abs((*A)[i+j*(A->ld())]-(*C)[i+j*(C->ld())]),
                        1e-12*normInf);
        }
    }
}

#if 0
TEST(TestCholesky, qrBigMatrix)
{

    INT m=1000;
    INT n=101;

    /*  A = [ 2 4;
     *        2 5;
     *        2 6;
     *      ]
     */
    std::shared_ptr<LinearAlgebra::Matrix<double>>
    A{new LinearAlgebra::Matrix<double>(m,n)};

    A->rand();

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    C{new LinearAlgebra::Matrix<double>(m,n)};

    auto iter = LinearAlgebra::max_element(A->begin(), A->end());
    double normInf = *(iter);

    for(uint i=0;i<m;i++)
    {
        for(uint j=0;j<n;j++)
        {
            EXPECT_NEAR(0.0,(*C)[i+j*(C->ld())],1e-12*normInf);
        }
    }

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    Q{new LinearAlgebra::Matrix<double>(m,n)};

    std::shared_ptr<LinearAlgebra::Matrix<double>>
    R{new LinearAlgebra::Matrix<double>(n,n)};

    *Q = *A;
    Orthogonalization::Cholesky<double> cholOrth(m,n);
    EXPECT_EQ(Orthogonalization::OrthogonalizationErr_t::NO_ERROR
              , cholOrth.QR(m,n,Q,R));

    *C = *Q;
    LinearAlgebra::Operation::trmm(LinearAlgebra::Operation::Layout::ColMajor,
                                    LinearAlgebra::Operation::Side::Right,
                                    LinearAlgebra::Operation::Uplo::Upper,
                                    LinearAlgebra::Operation::Op::NoTrans,
                                    LinearAlgebra::Operation::Diag::NonUnit,
                                    m,n,1.0,R->data(), R->ld(),C->data(),C->ld());


    for(uint i=0;i<m;i++)
    {
        for(uint j=0;j<n;j++)
        {
            EXPECT_NEAR(0,(*A)[i+j*(A->ld())]-(*C)[i+j*(C->ld())],
                        1e-12*normInf);
        }
    }
}
#endif
